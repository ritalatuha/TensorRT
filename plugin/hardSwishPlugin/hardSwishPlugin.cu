#include "hip/hip_runtime.h"
 #include "hardSwishPlugin.h"
 #include <hip/hip_fp16.h>


 template <typename T_DATA>
     __global__ void kernelCopy(
         int N,
         T_DATA* inputs,
         T_DATA* outputs
         )
 {
     int index = blockIdx.x * blockDim.x + threadIdx.x;
     if (index < N){
         outputs[index] = inputs[index];
     }
     __syncthreads();
 }

 __global__ void hswishkernel(
    const int n,
    const float* x,
    float* y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x)
    {
        float temp = 0;
        if (x[i] < -3.0) {
            temp = 0.0;
        } else if (x[i] > 3.0) {
            temp = x[i];
        } else {
            temp = x[i] * (x[i] + 3.0) / 6.0;
        }
        y[i] = temp;
    }
}

 template <typename T>
 int hardSwishInference(
     int batchSize,
     int iC,
     int iH,
     int iW,
     T* inputs,
     T* outputs,
     hipStream_t stream){
         // NCHW
         const int nThreads = 512;
         int lenCopy = iC * iH * iW;

         int nBlocksCopy = (int)((float)lenCopy / nThreads) + 1;

         for(int i=0; i < batchSize; ++i){
             // NOTE: kernelCopy kernel can be replaced with hipMemcpy function
             kernelCopy<<<nBlocksCopy, nThreads, 0, stream>>>(lenCopy, inputs, outputs);
             outputs += lenCopy;

             hswishkernel<<<nBlocksCopy, nThreads, 0, stream>>>(lenCopy,  inputs, outputs);
             outputs += lenCopy;
             inputs += lenCopy;
         }

     hipError_t err = hipGetLastError();
     if ( hipSuccess != err )
     {
         fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 __FILE__, __LINE__, hipGetErrorString( err ) );
         return 1;
     }
     return 0;
 }

 int HardSwishPlugin::enqueue(int batchSize, const void* const* inputs, void** outputs, void* workspace, hipStream_t stream)
 {

     return hardSwishInference(batchSize, iC, iH, iW, (float*)inputs[0], (float*)outputs[0], stream);
 }
